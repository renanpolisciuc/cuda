#include "hip/hip_runtime.h"
#include<stdio.h>
#include<stdlib.h>
#define N 35000
#define BLOCK_DIM_SIZE 32
#define MAX_INT_GEN 50
#define EXIT_SUCESS 0
#define EXIT_ERROR -1

void hostProGerarRandomNumbers(int * v, unsigned int n) {
  if (v == NULL || n == 0)
    exit(EXIT_ERROR);

  for(int i = 0; i < n; ++i) 
    v[i] = rand() % (MAX_INT_GEN);
}

__global__ void devProAddVectors(int * va, int * vb, int * vc, int n) {
  //Id da thread no vetor global
  int id = blockIdx.x * blockDim.x + threadIdx.x;

  //Verifica se o id é menor que o tamanho do vetor e soma
  if (id < n)
    vc[id] = va[id] + vb[id];
}


int main(int argc, char **argv) {
  //Pointers de vetor do host
  int *ha, *hb, *hc;

  //Pointers de vetor do device(placa)
  int *da, *db, *dc;

  //Quantidade de bytes alocados
  unsigned int bytesToAlloc = sizeof(int) * N;

  //Alocação dos vetores no host
  ha = (int *) malloc(bytesToAlloc);
  hb = (int *) malloc(bytesToAlloc);
  hc = (int *) malloc(bytesToAlloc);

  //Alocação dos vetores no device
  hipMalloc((void**)&da, bytesToAlloc);
  hipMalloc((void**)&db, bytesToAlloc);
  hipMalloc((void**)&dc, bytesToAlloc);

  //man srand
  srand(time(NULL));
  hostProGerarRandomNumbers(ha, N);
  hostProGerarRandomNumbers(hb, N);
  hostProGerarRandomNumbers(hc, N);

  //Cópia da memória do host pro device (os dados serão operados na memória do device)
  hipMemcpy(da, ha, bytesToAlloc, hipMemcpyHostToDevice);
  hipMemcpy(db, hb, bytesToAlloc, hipMemcpyHostToDevice);

  //Alocação dos blocos de threads
  // 32 * 32 = 1024 (threads em paralelo)
  // N / blkSize.x = quantidade de blocos na dimensão X
  // N / blkSize.x = quantidade de blocos na dimensão Y
  dim3 blkSize(BLOCK_DIM_SIZE, BLOCK_DIM_SIZE);
  dim3 numBlks((N / blkSize.x) + 1, (N /blkSize.y) + 1);

  //Chamada do kernel <<<Número de blocos, tamanho de cada bloco>>>
  devProAddVectors<<<numBlks, blkSize>>>(da, db, dc, N);

  //Copia memória processada do device para o host
  hipMemcpy(hc, dc, bytesToAlloc, hipMemcpyDeviceToHost);

  int ok = 1;
  int vEsperado, vObtido, sI;
  for(int i = 0; i < N; i++) {
    sI = ha[i] + hb[i]; 

    if (sI != hc[i]) {
      ok = 0;
      vEsperado = sI;
      vObtido = hc[i];
      break;
    }
  }

  if (!ok) 
    printf("Solução incorreta. Valor esperado = %d, valor obtido = %d\n", vEsperado, vObtido);
  else
    printf("Solução correta.\n");

  hipFree(da);
  hipFree(db);
  hipFree(dc);
  free(ha);
  free(hb);
  free(hc);

  return EXIT_SUCESS;
}
